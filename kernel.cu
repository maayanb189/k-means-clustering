#include "hip/hip_runtime.h"
#include "Kernel.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include <math.h>


#ifdef PARALLEL
int getNumOfBlock(int pointsArrSize, hipDeviceProp_t prop);
void checkErrorStatus(hipError_t e);

__device__ double calcDistanceCuda(double location1[DIMENSIONS], double location2[DIMENSIONS])
{
	double powDistanceSum = 0;
	int dimensionIndex;
	for (dimensionIndex = 0; dimensionIndex < DIMENSIONS; dimensionIndex++)
	{
		powDistanceSum += pow((location1[dimensionIndex] - location2[dimensionIndex]), 2);
	}
	double sqrtD = sqrt(powDistanceSum);
	return sqrtD;
}

__global__ void updatePointsLocation(Point * points, int numOfPoints, double timeInterval, int numOfThreadsPerBlock)
{
	int pointIndex = blockIdx.x*numOfThreadsPerBlock + threadIdx.x;
	if (pointIndex < numOfPoints)
	{
		int axisIndex;
		for (axisIndex = 0; axisIndex < DIMENSIONS; axisIndex++)
		{
			points[pointIndex].location[axisIndex] += timeInterval*points[pointIndex].velocity[axisIndex];
		}
	}
}

__global__ void groupPoints(Cluster * clusters, int numOfClusters, Point ** points, int numOfPoints, int numOfThreadsPerBlock, bool * pointsMoved)
{
	//*pointsMoved = false;
	int pointIndex = blockIdx.x*numOfThreadsPerBlock + threadIdx.x;

	if (pointIndex < numOfPoints)
	{
		Point * currentPoint = &((*points)[pointIndex]);

		int clusterIndex = 0;
		double minDistance = calcDistanceCuda(currentPoint->location, clusters[clusterIndex].center);
		int closestClusterIndex = clusterIndex;
		for (clusterIndex = 1; clusterIndex < numOfClusters; clusterIndex++)
		{
			double distance = calcDistanceCuda(currentPoint->location, clusters[clusterIndex].center);
			if (distance < minDistance)
			{
				minDistance = distance;
				closestClusterIndex = clusterIndex;
			}
		}
		//update the current cluster
		if (currentPoint->currentCluster != closestClusterIndex)
		{
			currentPoint->currentCluster = closestClusterIndex;
			*pointsMoved = true;
		}
	}
}


Point * allocatePointsOnGpuCuda(Point * points, int numOfPoints)
{
	Point * pointsGPU;
	hipError_t cudaStatus;
	
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	checkErrorStatus(cudaStatus);

	// Allocate GPU buffers for points
	cudaStatus = hipMalloc((void**)&pointsGPU, numOfPoints * sizeof(Point));
	checkErrorStatus(cudaStatus);

	//cuda memcpy to GPU
	cudaStatus = hipMemcpy(pointsGPU, points, numOfPoints * sizeof(Point), hipMemcpyHostToDevice);
	checkErrorStatus(cudaStatus);

	return pointsGPU;
}

Cluster * allocateClustersOnGPU(Cluster * clusters)
{
	Cluster * clustersGPU;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	checkErrorStatus(cudaStatus);

	// Allocate GPU buffers for clusters
	cudaStatus = hipMalloc((void**)&clustersGPU, params.numOfClusters * sizeof(Cluster));
	checkErrorStatus(cudaStatus);

	//cuda memcpy to GPU
	cudaStatus = hipMemcpy(clustersGPU, clusters, params.numOfClusters * sizeof(Cluster), hipMemcpyHostToDevice);
	checkErrorStatus(cudaStatus);

	return clustersGPU;
}

Point * progressPointsLocationCuda(Point * points, int numOfPoints, Point * pointArr_onGPU)
{
	hipError_t cudaStatus;
	hipDeviceProp_t prop;
	int numOfBlocks;
	//int pointIndex;

	//get device properties
	cudaStatus = hipGetDeviceProperties(&prop, 0);
	checkErrorStatus(cudaStatus);

	numOfBlocks = getNumOfBlock(numOfPoints, prop);

	// Choose which GPU to run on
	cudaStatus = hipSetDevice(0);
	checkErrorStatus(cudaStatus);

	updatePointsLocation << <numOfBlocks, prop.maxThreadsPerBlock >> >(pointArr_onGPU, numOfPoints, params.timeInterval, prop.maxThreadsPerBlock);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	checkErrorStatus(cudaStatus);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	checkErrorStatus(cudaStatus);

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(points, pointArr_onGPU, numOfPoints * sizeof(Point), hipMemcpyDeviceToHost);
	checkErrorStatus(cudaStatus);

//	for (pointIndex = 0; pointIndex < numOfPoints; pointIndex++)
//	{
//		cudaStatus = hipMemcpy(points->location, pointArr_onGPU->location, sizeof(points->location), hipMemcpyDeviceToHost);
//		checkErrorStatus(cudaStatus);
//	}
	return pointArr_onGPU;
}

//returns true if points have moved, else - false
bool groupPointsCuda(Cluster * clusters, Point ** pointsOnGPU, int numOfPoints)
{
	bool ret = false;
	hipError_t cudaStatus;
	hipDeviceProp_t prop;

	int numOfBlocks = 0;
	Cluster* clustersOnGPU;

	//get device properties
	cudaStatus = hipGetDeviceProperties(&prop, 0);
	checkErrorStatus(cudaStatus);

	cudaStatus = hipSetDevice(0);
	checkErrorStatus(cudaStatus);

	clustersOnGPU = allocateClustersOnGPU(clusters);

	numOfBlocks = getNumOfBlock(numOfPoints, prop);
	printf("numOfthreads : %d , numOfBlocks : %d\n", prop.maxThreadsPerBlock , numOfBlocks);
	fflush(stdout);

	
	ret = false;
	groupPoints << <numOfBlocks*2, prop.maxThreadsPerBlock/2 >> > (clustersOnGPU, params.numOfClusters, pointsOnGPU, numOfPoints, prop.maxThreadsPerBlock/2, &ret);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	checkErrorStatus(cudaStatus);

	
	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	checkErrorStatus(cudaStatus);

	
	//free cuda clusters
	hipFree(clustersOnGPU);

	
	return ret;
}


int getNumOfBlock(int numOfPoints, hipDeviceProp_t prop)
{
	int numOfBlocks = numOfPoints / prop.maxThreadsPerBlock;
	if (numOfPoints % prop.maxThreadsPerBlock)
	{
		numOfBlocks++;
	}
	return numOfBlocks;
}

void checkErrorStatus(hipError_t e)
{
	// check if the status from the cuda was ok
	if (e != hipSuccess)
	{
		printf("Cuda Error: %d\n", e);
		fflush(stdout);
		exit(1);
	}
}

#endif

